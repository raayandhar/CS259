#include <hipDNN.h>
#include <torch/extension.h>
#include <hip/hip_fp16.h>
#include <tuple>

#define checkCUDNN(expression)                             \
  {                                                        \
    hipdnnStatus_t status = (expression);                   \
    if (status != HIPDNN_STATUS_SUCCESS) {                  \
      std::cerr << "Error on line " << __LINE__ << ": "    \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                             \
    }                                                      \
  }

// Conv2d Layer 1 with cuDNN
std::tuple<torch::Tensor, float> launch_conv2d_cudnn_v1(torch::Tensor input, torch::Tensor filters, torch::Tensor output) {
    constexpr int in_channels = 64; // Ni
    constexpr int in_height = 224; // Ny
    constexpr int in_width = 224; // Nx
    
    constexpr int out_channels = 64; // Nn
    constexpr int kernel_height = 3; // Ky
    constexpr int kernel_width = 3; // Kx
    
    int stride_height = 1;
    int stride_width = 1;
    int pad_height = 0;
    int pad_width = 0;
    
    hipSetDevice(0);
    
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);
    
    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                         HIPDNN_TENSOR_NCHW,
                                         HIPDNN_DATA_HALF,
                                         1,
                                         in_channels,
                                         in_height,
                                         in_width));
    
    hipdnnFilterDescriptor_t kernel_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                         HIPDNN_DATA_HALF,
                                         HIPDNN_TENSOR_NCHW,
                                         out_channels,
                                         in_channels,
                                         kernel_height,
                                         kernel_width));
    
    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                              pad_height,
                                              pad_width,
                                              stride_height,
                                              stride_width,
                                              1,
                                              1,
                                              HIPDNN_CROSS_CORRELATION,
                                              HIPDNN_DATA_FLOAT));
    
 
//    checkCUDNN(hipdnnSetConvolutionMathType(convolution_descriptor, HIPDNN_TENSOR_OP_MATH));
    
    int batch_size{0}, channels{0}, height{0}, width{0};
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor,
                                                    input_descriptor,
                                                    kernel_descriptor,
                                                    &batch_size,
                                                    &channels,
                                                    &height,
                                                    &width));
    
    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                         HIPDNN_TENSOR_NCHW,
                                         HIPDNN_DATA_HALF,
                                         batch_size,
                                         channels,
                                         height,
                                         width));
    
    hipdnnConvolutionFwdAlgo_t convolution_algorithm = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
    
    size_t workspace_bytes{0};
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                     input_descriptor,
                                                     kernel_descriptor,
                                                     convolution_descriptor,
                                                     output_descriptor,
                                                     convolution_algorithm,
                                                     &workspace_bytes));
    
    void* workspace{nullptr};
    if (workspace_bytes > 0) {
        hipMalloc(&workspace, workspace_bytes);
    }
    
    const float alpha = 1.0f, beta = 0.0f;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    checkCUDNN(hipdnnConvolutionForward(cudnn,
                                     &alpha,
                                     input_descriptor,
                                     input.data_ptr<at::Half>(),
                                     kernel_descriptor,
                                     filters.data_ptr<at::Half>(),
                                     convolution_descriptor,
                                     convolution_algorithm,
                                     workspace,
                                     workspace_bytes,
                                     &beta,
                                     output_descriptor,
                                     output.data_ptr<at::Half>()));
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    if (workspace) {
        hipFree(workspace);
    }
    
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
    
    hipdnnDestroy(cudnn);
    
    return std::make_tuple(output, milliseconds);
}

// Conv2d Layer 2 with cuDNN
std::tuple<torch::Tensor, float> launch_conv2d_cudnn_v2(torch::Tensor input, torch::Tensor filters, torch::Tensor output) {
    constexpr int in_channels = 512; // Ni
    constexpr int in_height = 14; // Ny
    constexpr int in_width = 14; // Nx
    
    constexpr int out_channels = 512; // Nn
    constexpr int kernel_height = 3; // Ky
    constexpr int kernel_width = 3; // Kx
    
    int stride_height = 1;
    int stride_width = 1;
    int pad_height = 0;
    int pad_width = 0;
    
    hipSetDevice(0);
    
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);
    
    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                         HIPDNN_TENSOR_NCHW,
                                         HIPDNN_DATA_HALF,
                                         1,
                                         in_channels,
                                         in_height,
                                         in_width));
    
    hipdnnFilterDescriptor_t kernel_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                         HIPDNN_DATA_HALF,
                                         HIPDNN_TENSOR_NCHW,
                                         out_channels,
                                         in_channels,
                                         kernel_height,
                                         kernel_width));
    
    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                              pad_height,
                                              pad_width,
                                              stride_height,
                                              stride_width,
                                              1,
                                              1,
                                              HIPDNN_CROSS_CORRELATION,
                                              HIPDNN_DATA_FLOAT));
    
 
//    checkCUDNN(hipdnnSetConvolutionMathType(convolution_descriptor, HIPDNN_TENSOR_OP_MATH));
    
    int batch_size{0}, channels{0}, height{0}, width{0};
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor,
                                                    input_descriptor,
                                                    kernel_descriptor,
                                                    &batch_size,
                                                    &channels,
                                                    &height,
                                                    &width));
    
    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                         HIPDNN_TENSOR_NCHW,
                                         HIPDNN_DATA_HALF,
                                         batch_size,
                                         channels,
                                         height,
                                         width));
    
    hipdnnConvolutionFwdAlgo_t convolution_algorithm = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED;
    
    size_t workspace_bytes{0};
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                     input_descriptor,
                                                     kernel_descriptor,
                                                     convolution_descriptor,
                                                     output_descriptor,
                                                     convolution_algorithm,
                                                     &workspace_bytes));
    
    void* workspace{nullptr};
    if (workspace_bytes > 0) {
        hipMalloc(&workspace, workspace_bytes);
    }
    
    const float alpha = 1.0f, beta = 0.0f;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    checkCUDNN(hipdnnConvolutionForward(cudnn,
                                     &alpha,
                                     input_descriptor,
                                     input.data_ptr<at::Half>(),
                                     kernel_descriptor,
                                     filters.data_ptr<at::Half>(),
                                     convolution_descriptor,
                                     convolution_algorithm,
                                     workspace,
                                     workspace_bytes,
                                     &beta,
                                     output_descriptor,
                                     output.data_ptr<at::Half>()));
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    if (workspace) {
        hipFree(workspace);
    }
    
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
    
    hipdnnDestroy(cudnn);
    
    return std::make_tuple(output, milliseconds);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("conv2d_cudnn_v1", &launch_conv2d_cudnn_v1, "cuDNN convolution for layer 1");
    m.def("conv2d_cudnn_v2", &launch_conv2d_cudnn_v2, "cuDNN convolution for layer 2");
}
