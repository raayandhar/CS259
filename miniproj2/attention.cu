#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

/*
Forward kernel for multi-headed flash attention.
Q: input tensor of shape (batch_size, num_heads, seq_len, d_query)
K: input tensor of shape (batch_size, num_heads, seq_len, d_query)
V: input tensor of shape (batch_size, num_heads, seq_len, d_value)
O: output tensor of shape (batch_size, num_heads, seq_len, d_value)
l: row-wise exponential sum (used for softmax)
m: row-maximum (used for softmax)
B: batch size
H: number of heads
*/
__global__ void forward_kernel(const float* Q, const float* K, const float* V, float* O, float* l, float* m,
                    const int B, const int H, const int N, const int d,
                    const int Tc, const int Tr, const int Bc, const int Br, const float innerprod_scale) {

    /*
    b: batch index
    h: head index (in multi-head attention)
    threadIdx.x (tx): iterating over rows of a Q-tile
    threadIdx.y (ty): iterating over rows of a K/V-tile
    gridDim.x (bz): batch size
    gridDim.y (nh): total # of heads

    Parallelism scheme:
    Each block handles the attention computation for a specific batch element and a specific attention head.
    The local computation we deal with is Q[batch_idx][head_idx] @ K[batch_idx][head_idx]^T.

    Let's denote this as Q_i @ K_i^T, where Q_i has shape (seq_len, d_query) and K_i has shape (seq_len, d_query). This reduces
    to a matmul followed by a normalization operation (/sqrt(d_query)) and a softmax before a matmul with V_i:

    A_i (seq_len, seq_len) @ V_i (seq_len, d_value) => O_i (seq_len, d_value)
    */
    int b = blockIdx.x;    
    int h = blockIdx.y;    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bz = gridDim.x;    
    int nh = gridDim.y;    

    extern __shared__ float sram[]; // size specified in <<gridDim, blockDim, sharedMemSize>>

    /*
    Shared memory layout:
    [Q_tile (Br, query_dim), K_tile (Bc, query_dim), V_tile (Bc, query_dim), Sij (Br, Bc)]

    We compute a small partial product of the final attention pattern per thread. Br queries * Bc keys of it, in fact.
    We store all of this data contiguously the shared memory.
    */
    float* Qi = sram; // tile of Q matrix
    float* Kj = &sram[Br*d]; // tile of K matrix
    float* Vj = &sram[Br*d + Bc*d]; // tile of V matrix
    float* Sij = &sram[2*Br*d + 2*Bc*d + 2*Br]; // Q_i * K_j^T
    const int bnhNd = b * nh * N * d;
    const int hNd = h * N * d;

    /*
    Legacy
    //float* Oi = &sram[Br*d + 2*Bc*d];
    //float* li = &sram[2*Br*d + 2*Bc*d];
    //float* mi = &sram[2*Br*d + 2*Bc*d + Br];
    //float* Sij = &sram[Br*d + 2*Bc*d];
    */

    for(int j = 0; j < Tc; j++) {
        // Load Kj, Vj into shared memory
        for(int k = 0; k < d; k++){
            Kj[ty*d + k] = K[bnhNd + hNd + j*Bc*d + (ty*d + k)];
            Vj[ty*d + k] = V[bnhNd + hNd + j*Bc*d + (ty*d + k)];
        }
        __syncthreads();
        
        // Load Qi into shared memory
        for(int i = 0; i < Tr; i++) {
            for(int k = 0; k < d; k++){
                Qi[tx*d + k] = Q[bnhNd + hNd + i*Br*d + (tx*d + k)];
            }
            __syncthreads();

            // Load l, m into shared memory
            float li_old = l[b*nh*N + h*N + i*Br + tx];
            float mi_old = m[b*nh*N + h*N + i*Br + tx];

            // Compute QK^T
            float inner_prod = 0.0f;
            for(int k = 0; k < d; k++){
                inner_prod += Qi[tx*d + k] * Kj[ty*d + k];
            }
            // inner_prod *= innerprod_scale;
            Sij[tx*Bc + ty] = inner_prod;
            __syncthreads();

            // Compute mij 
            float mij = -INFINITY;

            
            for(int jj = 0; jj < Bc; jj++) {
                mij = fmaxf(mij, Sij[tx*Bc + jj]);
            }
            __syncthreads();
            

            // Compute Pij (safe softmax)
            Sij[tx*Bc + ty] = __expf(Sij[tx*Bc + ty] - mij);
            __syncthreads();


            float lij = 0.0f;
            for(int jj = 0; jj < Bc; jj++){
                float val = Sij[tx*Bc+jj];
                lij += val;
            }
            
            __syncthreads();
            
            // Compute mi_new, li_new
            float mi_new = max(mi_old, mij);
            float li_new = __expf(mi_old - mi_new) * li_old + __expf(mij - mi_new) * lij;

            // Write to O
            for(int k = 0; k < d; k++){
                float PijVj = 0.0f;
                for(int jj = 0; jj < Bc; jj++){
                    PijVj += Sij[tx*Bc + jj] * Vj[jj*d + k];
                }

                if(i*Br*d + tx*d < N*d){
                    if(ty == 0){ // make sure ty don't override each other
                    O[bnhNd + hNd + i*Br*d + (tx*d + k)] = 
                        (1.0f / li_new) * (li_old * expf(mi_old - mi_new) * O[bnhNd + hNd + i*Br*d + (tx*d + k)] + 
                                        expf(mij - mi_new) * PijVj);
                    }
                    // Write to l, m
                    l[b*nh*N + h*N + i*Br + tx] = li_new;
                    m[b*nh*N + h*N + i*Br + tx] = mi_new;
                }
            }
        }
        __syncthreads();
    }
}

torch::Tensor forward(torch::Tensor Q, torch::Tensor K, torch::Tensor V, torch::Tensor O, torch::Tensor l, torch::Tensor m, const int bz, const int nh, const int N, const int d) {
    int Br = 16;
    int Bc = 16;

    const int Tr = (N + Br - 1) / Br;
    const int Tc = (N + Bc - 1) / Bc;
    const float dot_prod_scale = 1.0f/sqrt(d);

    dim3 grid_size(bz, nh);
    dim3 block_size(Br, Bc);
    const int shared_mem_size = (2*Br*d + 2*Bc*d + 2*Br + Br*Bc) * sizeof(float);

    forward_kernel<<<grid_size, block_size, shared_mem_size>>>(
        Q.data_ptr<float>(),
        K.data_ptr<float>(),
        V.data_ptr<float>(),
        O.data_ptr<float>(),
        l.data_ptr<float>(),
        m.data_ptr<float>(),
        bz, nh, N, d, Tc, Tr, Bc, Br, dot_prod_scale
    );
    return O;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Flash Attention forward");
}


